#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <VoiceUtil.cuh>

#include <vector>

#ifndef max
#define max(a,b)            (((a) > (b)) ? (a) : (b))
#endif

#ifndef min
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#endif

template <class T>
class CUDAVector
{
public:
	unsigned count;
	T* d_data;
};

template <class T_GPU, class T_CPU>
class CUDAImagedVector : public CUDAVector<T_GPU>{};

template <class T>
class CUDALevel2Vector : public CUDAImagedVector<CUDAVector<T>, std::vector<T>>{};

class CUDASrcBuf : public CUDAVector<float> {};

struct SourceInfo;
typedef CUDAImagedVector<CUDASrcBuf, SourceInfo> CUDASrcBufList;

struct SrcSampleInfo
{
	unsigned srcPos;
	float srcSampleFreq;
	float logicalPos;
};

struct CUDASrcPieceInfo
{
	CUDAVector<SrcSampleInfo> SampleLocations;
	CUDAVector<SrcSampleInfo> SampleLocations_next;
	unsigned fixedBeginId;
	unsigned fixedEndId;
	unsigned fixedBeginId_next;
	unsigned fixedEndId_next;
};

struct SrcPieceInfo;
typedef CUDAImagedVector<CUDASrcPieceInfo, SrcPieceInfo> CUDASrcPieceInfoList;

struct Job
{
	unsigned pieceId;
	unsigned isNext;
	unsigned jobOfPiece;
};


__shared__ unsigned char sbuf[];

__global__
void g_GetMaxVoiced(CUDASrcBufList cuSrcBufs, CUDASrcPieceInfoList pieceInfoList,
CUDALevel2Vector<unsigned> cuMaxVoicedLists, CUDALevel2Vector<unsigned> cuMaxVoicedLists_next, CUDAVector<Job> jobMap, unsigned BufSize)
{
	const Job& job = jobMap.d_data[blockIdx.x];
	const CUDASrcPieceInfo& pieceInfo = pieceInfoList.d_data[job.pieceId];
	bool isNext = job.isNext != 0;
	unsigned paramId = job.jobOfPiece + isNext ? pieceInfo.fixedBeginId_next : pieceInfo.fixedBeginId;

	unsigned numWorker = blockDim.x;
	unsigned workerId = threadIdx.x;

	SrcSampleInfo& posInfo = isNext ? pieceInfo.SampleLocations_next.d_data[paramId] : pieceInfo.SampleLocations.d_data[paramId];

	float fhalfWinlen = 3.0f / posInfo.srcSampleFreq;
	unsigned u_halfWidth = (unsigned)ceilf(fhalfWinlen);
	unsigned uSpecLen = (unsigned)ceilf(fhalfWinlen*0.5f);

	float *s_buf1 = (float*)sbuf;
	float *s_buf2 = (float*)sbuf + u_halfWidth * 2;

	const CUDASrcBuf& srcBuf = cuSrcBufs.d_data[job.pieceId];

	d_captureFromBuf(srcBuf.count, srcBuf.d_data, posInfo.srcPos, fhalfWinlen, u_halfWidth, s_buf1);
	d_CreateAmpSpectrumFromWindow(fhalfWinlen, u_halfWidth, s_buf1, s_buf2, uSpecLen);

	unsigned& maxVoiced = *((unsigned*)sbuf + BufSize - 1);
	maxVoiced = 0;

	__syncthreads();

	for (unsigned i = 6 + 3 * workerId; i + 4 < uSpecLen; i += 3 * numWorker)
	{
		unsigned count = 0;
		for (int j = -3; j <= 3; j += 3)
		{
			float absv0 = s_buf2[(int)i + j];
			float absv1 = s_buf2[(int)i + j - 1];
			float absv2 = s_buf2[(int)i + j + 1];

			float rate = absv0 / (absv0 + absv1 + absv2);

			if (rate > 0.7f)
			{
				count++;
			}
		}
		if (count > 1)
		{
			atomicMax(&maxVoiced, i / 3);
		}
	}

	__syncthreads();

	CUDAVector<unsigned>& d_maxVoiced = isNext ? cuMaxVoicedLists.d_data[job.pieceId] : cuMaxVoicedLists_next.d_data[job.pieceId];

	if (workerId==0)
		d_maxVoiced.d_data[job.jobOfPiece] = maxVoiced;
}

void h_GetMaxVoiced(CUDASrcBufList cuSrcBufs, CUDASrcPieceInfoList pieceInfoList,
	CUDALevel2Vector<unsigned> cuMaxVoicedLists, CUDALevel2Vector<unsigned> cuMaxVoicedLists_next, CUDAVector<Job> jobMap, unsigned BufSize)
{
	static const unsigned groupSize = 256;
	unsigned sharedBufSize = (unsigned)sizeof(float)* BufSize;
	g_GetMaxVoiced << < jobMap.count, groupSize, sharedBufSize >> > (cuSrcBufs, pieceInfoList, cuMaxVoicedLists, cuMaxVoicedLists_next, jobMap, BufSize);
}
